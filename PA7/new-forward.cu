#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16

__global__ void conv_forward_kernel(float *y, const float * __restrict__ x, const float * __restrict__ k, const int B, const int M, const int C, const int H, const int W, const int K)
{

    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.
    We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    */

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    //(void)H_out; // silence declared but never referenced warning. remove this line when you start working
    //(void)W_out; // silence declared but never referenced warning. remove this line when you start working

// An example use of these macros:
// float a = y4d(0,0,0,0)
// y4d(0,0,0,0) = a
#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

     int b = blockIdx.x, m = blockIdx.y;

     int horizontalTiles = ceil((W_out)*1.0/TILE_WIDTH);

     int h = (blockIdx.z / horizontalTiles)*TILE_WIDTH + threadIdx.y, w = (blockIdx.z % horizontalTiles)*TILE_WIDTH+threadIdx.x;

     float acc = 0.0f;

     for(int c = 0; c < C; c++) {
         for(int r = 0; r < K; r++) {
             for(int col = 0; col < K; col++) {
                 acc += x4d(b, c, h+r, w+col) * k4d(m, c, r, col);
             }
         }
     }

     if(h < H_out && w < W_out)
        y4d(b, m, h, w) = acc;

#undef y4d
#undef x4d
#undef k4d
}



__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_y, const float *host_x, const float *host_k, float **device_y_ptr, float **device_x_ptr, float **device_k_ptr, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU
    hipMalloc(device_x_ptr, B*C*H*W*sizeof(float));
    hipMalloc(device_y_ptr, B*M*(W-K+1)*(H-K+1)*sizeof(float));
    hipMalloc(device_k_ptr, M*C*K*K*sizeof(float));

    hipMemcpy(*device_x_ptr, host_x, B*C*H*W*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_k_ptr, host_k, M*C*K*K*sizeof(float), hipMemcpyHostToDevice);

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_y, const float *device_x, const float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Set the kernel dimensions and call the kernel
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);

    int Z = (int)ceil((W-K+1)*1.0/TILE_WIDTH)*(int)ceil((H-K+1)*1.0/TILE_WIDTH);

    dim3 gridDim(B,M,Z);

    conv_forward_kernel<<<gridDim, blockDim>>>(device_y, device_x, device_k, B, M, C, H, W, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_y, float *device_y, float *device_x, float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Copy the output back to host
    hipMemcpy(host_y, device_y, B*M*(H-K+1)*(W-K+1)*sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_y);
    hipFree(device_x);
    hipFree(device_k);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}