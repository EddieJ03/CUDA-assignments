#include "hip/hip_runtime.h"
#include <gputk.h>
#include <hip/hip_runtime.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;

  if(threadId < len) {
    out[threadId] = in1[threadId] + in2[threadId];
  }
}

int main(int argc, char **argv) {
  gpuTKArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)gpuTKImport(gpuTKArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The input length is ", inputLength);

  gpuTKTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  int len = inputLength*sizeof(float);
  hipMalloc((void**)&deviceInput1, len);
  hipMalloc((void**)&deviceInput2, len);
  hipMalloc((void**)&deviceOutput, len);

  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, len, hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, len, hipMemcpyHostToDevice);

  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  float NUM_THREADS = 256.0;
  int NUM_BLOCKS = (int)ceil(inputLength / NUM_THREADS);

  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  vecAdd<<<NUM_BLOCKS, NUM_THREADS>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

  hipDeviceSynchronize();
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, len, hipMemcpyDeviceToHost);

  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  gpuTKSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}